#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    int index = threadIdx.x;
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int N = 1024; // Vector length
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    size_t size = N * sizeof(int);

    // Allocate memory for host arrays
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    // Initialize vectors a and b with sample values
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i;
    }

    // Allocate memory for device arrays
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch kernel with block size N
    vectorAdd<<<1, N>>>(d_a, d_b, d_c, N);

    // Copy result from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < N; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    // Free memory
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
