#include <stdio.h>
#include <hip/hip_runtime.h>

#define M 2
#define N 3

__global__ void row_col_sum_kernel(int *A, int *B, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        int sum = 0;
        for (int i = 0; i < cols; i++) sum += A[row * cols + i]; // Row sum
        for (int i = 0; i < rows; i++) sum += A[i * cols + col]; // Column sum
        B[row * cols + col] = sum;
    }
}

int main() {
    int A[M][N] = {{1, 2, 3}, {4, 5, 6}};
    int B[M][N] = {0};
    int *d_A, *d_B;

    // Allocate device memory
    hipMalloc((void **)&d_A, M * N * sizeof(int));
    hipMalloc((void **)&d_B, M * N * sizeof(int));

    // Copy data to device
    hipMemcpy(d_A, A, M * N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(3, 2);
    dim3 blocksPerGrid(1, 1);
    row_col_sum_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, M, N);

    // Copy result back to host
    hipMemcpy(B, d_B, M * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    printf("Output matrix B:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", B[i][j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);

    return 0;
}