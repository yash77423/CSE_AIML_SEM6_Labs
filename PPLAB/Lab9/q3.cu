#include <stdio.h>
#include <hip/hip_runtime.h>

#define M 4
#define N 4

__global__ void complement_kernel(int *A, int *B, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        if (row > 0 && row < rows - 1 && col > 0 && col < cols - 1) {
            B[row * cols + col] = ~A[row * cols + col]; // 1's complement
        } else {
            B[row * cols + col] = A[row * cols + col]; // Border remains same
        }
    }
}

int main() {
    int A[M][N] = {{1, 2, 3, 4}, {6, 5, 8, 3}, {2, 4, 10, 1}, {9, 1, 2, 5}};
    int B[M][N] = {0};
    int *d_A, *d_B;

    // Allocate device memory
    hipMalloc((void **)&d_A, M * N * sizeof(int));
    hipMalloc((void **)&d_B, M * N * sizeof(int));

    // Copy data to device
    hipMemcpy(d_A, A, M * N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(4, 4);
    dim3 blocksPerGrid(1, 1);
    complement_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, M, N);

    // Copy result back to host
    hipMemcpy(B, d_B, M * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    printf("Output matrix B:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", B[i][j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);

    return 0;
}