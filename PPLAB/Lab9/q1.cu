#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 4 // Size of the matrix and vector

__global__ void spmv_csr_kernel(int *row_ptr, int *col_idx, float *values, float *x, float *y, int num_rows) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        float dot = 0;
        int row_start = row_ptr[row];
        int row_end = row_ptr[row + 1];
        for (int i = row_start; i < row_end; i++) {
            dot += values[i] * x[col_idx[i]];
        }
        y[row] = dot;
    }
}

int main() {
    // Input sparse matrix in CSR format
    int row_ptr[] = {0, 2, 4, 7, 8};
    int col_idx[] = {0, 1, 1, 2, 0, 2, 3, 1};
    float values[] = {1, 2, 3, 4, 5, 6, 7, 8};
    float x[N] = {1, 2, 3, 4}; // Input vector
    float y[N] = {0}; // Output vector

    // Allocate device memory
    int *d_row_ptr, *d_col_idx;
    float *d_values, *d_x, *d_y;
    hipMalloc((void **)&d_row_ptr, (N + 1) * sizeof(int));
    hipMalloc((void **)&d_col_idx, 8 * sizeof(int));
    hipMalloc((void **)&d_values, 8 * sizeof(float));
    hipMalloc((void **)&d_x, N * sizeof(float));
    hipMalloc((void **)&d_y, N * sizeof(float));

    // Copy data to device
    hipMemcpy(d_row_ptr, row_ptr, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, col_idx, 8 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, 8 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    spmv_csr_kernel<<<1, N>>>(d_row_ptr, d_col_idx, d_values, d_x, d_y, N);

    // Copy result back to host
    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    printf("Result vector y:\n");
    for (int i = 0; i < N; i++) {
        printf("%f ", y[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}