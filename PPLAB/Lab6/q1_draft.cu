
#include <hip/hip_runtime.h>
__global__ void convolution_1D_basic_kernel(float *N, float *M, float *P, int Mask_Width, int Width){
  
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  float Pvalue = 0;
  int N_start_point = i - (Mask_Width/2);
  for(int j = 0; j < Mask_Width; j++){
    if(N_start_point + j >= 0 && N_start_point + j < Width){
      Pvalue += N[N_start_point + j]*M[j];
    }
  }
  P[i] = Pvalue;
  
}
