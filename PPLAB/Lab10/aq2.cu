#include <stdio.h>
#include <hip/hip_runtime.h>

#define WIDTH 8
#define MASK_WIDTH 3
#define TILE_WIDTH 4

__global__ void tiledConvolution1D(int *N, int *M, int *P, int width) {
    __shared__ int N_s[TILE_WIDTH + MASK_WIDTH - 1];
    int tid = threadIdx.x;
    int offset = blockIdx.x * TILE_WIDTH;

    if (offset + tid < width) {
        N_s[tid] = N[offset + tid];
    } else {
        N_s[tid] = 0;
    }
    __syncthreads();

    if (offset + tid < width) {
        int sum = 0;
        for (int j = 0; j < MASK_WIDTH; j++) {
            if (tid + j < TILE_WIDTH + MASK_WIDTH - 1) {
                sum += N_s[tid + j] * M[j];
            }
        }
        P[offset + tid] = sum;
    }
}

int main() {
    int *N, *M, *P;
    int *d_N, *d_M, *d_P;
    int size_N = WIDTH * sizeof(int);
    int size_M = MASK_WIDTH * sizeof(int);

    N = (int*)malloc(size_N);
    M = (int*)malloc(size_M);
    P = (int*)malloc(size_N);

    printf("Enter input array (size %d):\n", WIDTH);
    for (int i = 0; i < WIDTH; i++) scanf("%d", &N[i]);

    printf("Enter mask (size %d):\n", MASK_WIDTH);
    for (int i = 0; i < MASK_WIDTH; i++) scanf("%d", &M[i]);

    hipMalloc(&d_N, size_N);
    hipMalloc(&d_M, size_M);
    hipMalloc(&d_P, size_N);

    hipMemcpy(d_N, N, size_N, hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, size_M, hipMemcpyHostToDevice);

    tiledConvolution1D<<<(WIDTH + TILE_WIDTH - 1) / TILE_WIDTH, TILE_WIDTH>>>(d_N, d_M, d_P, WIDTH);

    hipMemcpy(P, d_P, size_N, hipMemcpyDeviceToHost);

    printf("Convolution Result:\n");
    for (int i = 0; i < WIDTH; i++) printf("%d ", P[i]);
    printf("\n");

    hipFree(d_N); hipFree(d_M); hipFree(d_P);
    free(N); free(M); free(P);
    return 0;
}