#include <iostream>
#include <hip/hip_runtime.h>

#define N 4  // Matrix size (N x N)

__global__ void matrixMulRow(int *A, int *B, int *C, int width) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < width) {
        for (int col = 0; col < width; col++) {
            C[row * width + col] = 0;
            for (int k = 0; k < width; k++) {
                C[row * width + col] += A[row * width + k] * B[k * width + col];
            }
        }
    }
}

__global__ void matrixMulCol(int *A, int *B, int *C, int width) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < width) {
        for (int row = 0; row < width; row++) {
            C[row * width + col] = 0;
            for (int k = 0; k < width; k++) {
                C[row * width + col] += A[row * width + k] * B[k * width + col];
            }
        }
    }
}

__global__ void matrixMulElement(int *A, int *B, int *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < width && col < width) {
        C[row * width + col] = 0;
        for (int k = 0; k < width; k++) {
            C[row * width + col] += A[row * width + k] * B[k * width + col];
        }
    }
}

int main() {
    int A[N][N], B[N][N], C[N][N];
    int *d_A, *d_B, *d_C;

    // Initialize matrices A and B
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i][j] = i + j;
            B[i][j] = i - j;
        }
    }

    // Allocate memory on device
    hipMalloc((void**)&d_A, N * N * sizeof(int));
    hipMalloc((void**)&d_B, N * N * sizeof(int));
    hipMalloc((void**)&d_C, N * N * sizeof(int));

    // Copy matrices A and B to device
    hipMemcpy(d_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel (2a)
    matrixMulRow<<<1, N>>>(d_A, d_B, d_C, N);

    // Kernel launch (2b)
    // matrixMulCol<<<1, N>>>(d_A, d_B, d_C, N);

    // Kernel launch (2c)
    // dim3 threadsPerBlock(N, N);
    // matrixMulElement<<<1, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}